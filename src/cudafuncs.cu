#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


/* this is an explicit definition for atomicAdd, to be safe */
__device__ double atomicAdd(double* address, double val)
{
 unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do { assumed = old;
  old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed))); // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) 
  }
  while (assumed != old);
  return __longlong_as_double(old);
}


// minimal data to send to GPU. this is all that's needed to calc forces.
typedef struct atom_t {
    double pos[3]={0,0,0};
    double eps=0; // lj
    double sig=0; // lj
    double charge=0;
    double f[3]={0,0,0}; // force
    int molid=0;
} cuda_atom;

__global__
void calculateForceKernel(cuda_atom * atom_list, int N, double cutoff, double * basis, double * reciprocal_basis, int pform) {
    // define thread id
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    register cuda_atom anchoratom = atom_list[i];

    // PROBLEM WITH GLOBAL MEMORY ACCESS CONFLICT?


    // only run for real atoms (no ghost threads)
    if(i<N){   
        //printf("I AM THREAD %i\n", i);
        //atom_list[i].pos[0] += cutoff;
        register double rimg, rsq;
        double d[3], di[3], img[3], dimg[3],r,r2,ri,ri2;
        int p,q,j,n;
        double sig,eps,r6,s6,f[3]={0,0,0};//u[3];
        //int count=0;
        //register double af[3] = {0,0,0}; // accumulated forces
            //printf("basis[3] = %f\n", basis[3]);
        __syncthreads();
        // order N instead of N^2 bc this runs on all GPU cores at once (basically)
        for (j=i+1;j<N;j++) {

            if (anchoratom.molid == atom_list[j].molid) continue; // skip self-molecule interactions

            // get R (nearest image)
            for (n=0;n<3;n++) d[n] = anchoratom.pos[n] - atom_list[j].pos[n];
            for (p=0;p<3;p++) {
                img[p]=0;
                for (q=0;q<3;q++) {
                    img[p] += reciprocal_basis[p*3+q]*d[q];
                    //if (i==0 && j==1188) printf("img[%i] = reciprocal_basis[%i]*d[%i] = %f\n",p,p*3+q,q,reciprocal_basis[p*3+q]*d[q]);
                }
                img[p] = rint(img[p]);
            }
            for (p=0;p<3;p++) {
                di[p] = 0;
                for (q=0;q<3;q++) {
                    di[p] += basis[p*3+q]*img[q];
                }
            }
            for (p=0;p<3;p++) di[p] = d[p] - di[p];
            r2=0;ri2=0;
            for (p=0;p<3;p++) {
                r2 += d[p]*d[p];
                ri2 += di[p]*di[p];
            }
            r = sqrt(r2);
            ri = sqrt(ri2);
            if (ri != ri) {
                rimg=r;
                for (p=0;p<3;p++) dimg[p] = d[p];
            } else {
                rimg=ri;
                for (p=0;p<3;p++) dimg[p] = di[p];
            }
            // distance is now rimg
               
            //if (i==0) {
              //  printf("r[%i].%i = %f\n", i,j,rimg);
                //printf("CUTOFF: %f\n", cutoff);
                //for (int h=0;h<9;h++) {
                  //  printf("basis[%i] = %f\n", h, basis[h]);
                //}
            //}
            
            rsq=rimg*rimg;

            // 0 is LJ, 1 is LJ+ES
            if (pform == 0 || pform == 1) {
                //if (i==0) printf("hi\n");
                sig = anchoratom.sig;
                if (sig != atom_list[j].sig) sig = 0.5*(sig+atom_list[j].sig);
                eps = anchoratom.eps;
                if (eps != atom_list[j].eps) eps = sqrt(anchoratom.eps * atom_list[j].eps);

                if (sig == 0 || eps == 0) continue;

                r6 = rsq*rsq*rsq;
                s6 = sig*sig;
                s6 *= s6 * s6;

                if (rimg <= cutoff) {
                    for (n=0;n<3;n++) {
                        f[n] = 24.0*dimg[n]*eps*(2*(s6*s6)/(r6*r6*rsq) - s6/(r6*rsq));
                        atomicAdd(&(atom_list[j].f[n]), -f[n]);       
                        atomicAdd(&(atom_list[i].f[n]), f[n]);
                        //af[n] += f[n];
                    }
                    //if (i==0) count++;

                }

            }
            if (pform == 1) {
                //for (n=0;n<3;n++) u[n] = dimg[n]/r;
            }     


            //__syncthreads();
        } // end pair j

        //if (i==0) printf("COUNT: %i\n",count);
    } // end if i<n (all threads)
}

void CUDA_force(System &system) {

    const int N = (int)system.constants.total_atoms;
    const int block_size = system.constants.cuda_block_size; 
    const int atoms_array_size=sizeof(cuda_atom)*N;
    int index=0;

    cuda_atom H[N]; // host atoms
    cuda_atom *D; // device atoms (gpu)
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            H[index].molid = i;
            H[index].sig = system.molecules[i].atoms[j].sig;
            H[index].eps = system.molecules[i].atoms[j].eps;
            H[index].charge = system.molecules[i].atoms[j].C;
            for (int n=0;n<3;n++) {
                H[index].pos[n] = system.molecules[i].atoms[j].pos[n];       
                H[index].f[n] = 0; // initialize to zero
            }     
            index++;       
        }
    }

    int bs = sizeof(double)*9;
    double *basis;
    double *reciprocal_basis;
    basis = (double*)malloc(bs);
    reciprocal_basis = (double*)malloc(bs);
    double *dbasis;
    double *dreciprocal_basis;

    for (int p=0;p<3;p++) {
        for (int q=0;q<3;q++) {
            basis[3*q+p] = system.pbc.basis[p][q]; // quite sure correct.
            reciprocal_basis[3*q+p] = system.pbc.reciprocal_basis[p][q]; // quite sure correct.
        }
    }
    //system.pbc.printBasis();

    //for (int l=0;l<9;l++) printf("basis[%i] = %f\n", l,basis[l]);

    // allocate memory on GPU
    hipMalloc((void**) &dbasis, bs);
    hipMemcpy(dbasis, basis, bs, hipMemcpyHostToDevice);
    hipMalloc((void**) &dreciprocal_basis, bs);
    hipMemcpy(dreciprocal_basis, reciprocal_basis, bs, hipMemcpyHostToDevice); 
    hipMalloc((void**) &D, atoms_array_size);
    hipMemcpy(D, H, atoms_array_size, hipMemcpyHostToDevice);

    // grid elements
    int dimGrid = ceil((double)N/block_size);
    int dimBlock = block_size;   

    // assign potential form for force calculator
    int pform,theval=system.constants.potential_form;
    if (theval == POTENTIAL_LJ || theval == POTENTIAL_LJES || theval == POTENTIAL_LJESPOLAR)
        pform=0;
    if (theval == POTENTIAL_LJES || theval == POTENTIAL_LJESPOLAR)
        pform=1;

    calculateForceKernel<<< dimGrid, dimBlock >>>(D,N,system.pbc.cutoff, dbasis, dreciprocal_basis, pform);
    // make sure the threads are synced so we don't overflow
    hipDeviceSynchronize();
    // copy device data back to host
    hipMemcpy(H, D, atoms_array_size, hipMemcpyDeviceToHost);

    //for (int i=0;i<N;i++) printf("H[%i] force0 = %f\n", i, H[i].f[0]);
    index=0;
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            for (int n=0;n<3;n++) {
                system.molecules[i].atoms[j].force[n] = H[index].f[n];
            }     
            index++;       
        }
    }

    //printf("H[0] force = %f %f %f\n",system.molecules[0].atoms[0].force[0], system.molecules[0].atoms[0].force[1], system.molecules[0].atoms[0].force[2]);
 

     hipFree(D);

    // we're done. forces have been calc'd on GPU and written to local mem.
}
