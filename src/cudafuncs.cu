#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


/* this is an explicit definition for atomicAdd, to be safe */
__device__ double atomicAdd(double* address, double val)
{
 unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do { assumed = old;
  old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed))); // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) 
  }
  while (assumed != old);
  return __longlong_as_double(old);
}


// minimal data to send to GPU. this is all that's needed to calc forces.
typedef struct atom_t {
    double pos[3]={0,0,0};
    double eps=0; // lj
    double sig=0; // lj
    double charge=0;
    double f[3]={0,0,0}; // force
    int molid=0;
    int frozen=0;
} cuda_atom;

/*
// same but for molecule
typedef struct molecule_t {
    double old_ang_acc[3]={0,0,0};
    double ang_acc[3]={0,0,0};
    double ang_vel[3]={0,0,0};
    double ang_pos[3]={0,0,0};
    double torque[3]={0,0,0};
    double inertia=0;
    double mass=0;
    double old_acc[3]={0,0,0};
    double acc[3]={0,0,0};
    double vel[3]={0,0,0};
    double com[3]={0,0,0};
    double force[3]={0,0,0};
} cuda_molecule;
*/

__global__
void calculateForceKernel(cuda_atom * atom_list, int N, double cutoffD, double * basis, double * reciprocal_basis, int pformD, double ewald_alpha, int ewald_num_k, double kmax) {
    // define thread id
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // only run for real atoms (no ghost threads)
    if(i<N){   
        const register cuda_atom anchoratom = atom_list[i];
        //printf("I AM THREAD %i\n", i);
        //atom_list[i].pos[0] += cutoff;
       const int pform = pformD;
         const double alpha = ewald_alpha;
        const double cutoff=cutoffD;
        register double rimg, rsq;
        const double sqrtPI=sqrt(M_PI);
        double d[3], di[3], img[3], dimg[3],r,r2,ri,ri2;
        int q,j,n;
        double sig,eps,r6,s6,u[3]={0,0,0};
        //int count=0;
        register double af[3] = {0,0,0}; // accumulated forces for anchoratom
        double holder,chargeprod; // for ES force    
        //printf("basis[3] = %f\n", basis[3]);
        __syncthreads();
        // order N instead of N^2 bc this runs on all GPU cores at once (basically)

        // if LJ 
        if (pform == 0 || pform == 1) {
        for (j=i+1;j<N;j++) {

           if (anchoratom.molid == atom_list[j].molid) continue; // skip same molecule 
            if (anchoratom.frozen && atom_list[j].frozen) continue; // skip frozens            

           
            // get R (nearest image)
            for (n=0;n<3;n++) d[n] = anchoratom.pos[n] - atom_list[j].pos[n];
            for (n=0;n<3;n++) {
                img[n]=0;
                for (q=0;q<3;q++) {
                    img[n] += reciprocal_basis[n*3+q]*d[q];
                    //if (i==0 && j==1188) printf("img[%i] = reciprocal_basis[%i]*d[%i] = %f\n",p,p*3+q,q,reciprocal_basis[p*3+q]*d[q]);
                }
                img[n] = rint(img[n]);
            }
            for (n=0;n<3;n++) {
                di[n] = 0;
                for (q=0;q<3;q++) {
                    di[n] += basis[n*3+q]*img[q];
                }
            }
            for (n=0;n<3;n++) di[n] = d[n] - di[n];
            r2=0;ri2=0;
            for (n=0;n<3;n++) {
                r2 += d[n]*d[n];
                ri2 += di[n]*di[n];
            }
            r = sqrt(r2);
            ri = sqrt(ri2);
            if (ri != ri) {
                rimg=r;
                rsq=r2;
                for (n=0;n<3;n++) dimg[n] = d[n];
            } else {
                rimg=ri;
                rsq=ri2;
                for (n=0;n<3;n++) dimg[n] = di[n];
            }
            // distance is now rimg
               
            //if (i==0) {
              //  printf("r[%i].%i = %f\n", i,j,rimg);
                //printf("CUTOFF: %f\n", cutoff);
                //for (int h=0;h<9;h++) {
                  //  printf("basis[%i] = %f\n", h, basis[h]);
                //}
            //}

                if (rimg <= cutoff) {
           
                 sig = anchoratom.sig;
                if (sig != atom_list[j].sig) sig = 0.5*(sig+atom_list[j].sig);
                eps = anchoratom.eps;
                if (eps != atom_list[j].eps) eps = sqrt(eps * atom_list[j].eps);

                if (sig == 0 || eps == 0) continue;

     
                
                r6 = rsq*rsq*rsq;
                s6 = sig*sig;
                s6 *= s6 * s6;
        
                    for (n=0;n<3;n++) {
                        holder = 24.0*dimg[n]*eps*(2*(s6*s6)/(r6*r6*rsq) - s6/(r6*rsq));
                        atomicAdd(&(atom_list[j].f[n]), -holder); 
                        af[n] += holder;      
                    }
                }

        } // end pair j
        
        // finally add the accumulated forces (stored on register) to the anchor atom
        for (n=0;n<3;n++)
            atomicAdd(&(atom_list[i].f[n]), af[n]);
        
        } // end if LJ
        // ==============================================================================
        // Now handle electrostatics
        if (pform == 1) {
            for (n=0;n<3;n++) af[n]=0; // reset register-stored force for anchoratom.
          
                // get inverse volume
                double invV = basis[0]*(basis[4]*basis[8] - basis[7]*basis[5]);
                invV +=       basis[3]*(basis[7]*basis[2] - basis[1]*basis[8]);
                invV +=       basis[6]*(basis[1]*basis[5] - basis[5]*basis[2]);
                invV = 1.0/invV;
 
                double k_sq; double fourPI = M_PI*4;

        for (j=0;j<N;j++) {
                if (anchoratom.frozen && atom_list[j].frozen) continue; // don't do frozen pairs
                if (anchoratom.charge == 0 || atom_list[j].charge == 0) continue; // skip 0-force
                if (i==j) continue; // don't do atom with itself


               // get R (nearest image)
            for (n=0;n<3;n++) d[n] = anchoratom.pos[n] - atom_list[j].pos[n];
            for (n=0;n<3;n++) {
                img[n]=0;
                for (q=0;q<3;q++) {
                    img[n] += reciprocal_basis[n*3+q]*d[q];
                }
                img[n] = rint(img[n]);
            }
            for (n=0;n<3;n++) {
                di[n] = 0;
                for (q=0;q<3;q++) {
                    di[n] += basis[n*3+q]*img[q];
                }
            }
            for (n=0;n<3;n++) di[n] = d[n] - di[n];
            r2=0;ri2=0;
            for (n=0;n<3;n++) {
                r2 += d[n]*d[n];
                ri2 += di[n]*di[n];
            }
            r = sqrt(r2);
            ri = sqrt(ri2);
            if (ri != ri) {
                rimg=r;
                rsq=r2;
                for (n=0;n<3;n++) dimg[n] = d[n];
            } else {
                rimg=ri;
                rsq=ri2;
                for (n=0;n<3;n++) dimg[n] = di[n];
            }


            // real-space forces
            if (rimg <= cutoff && (anchoratom.molid < atom_list[j].molid)) { // non-duplicated pairs, not intramolecular, not beyond cutoff
                chargeprod = anchoratom.charge * atom_list[j].charge;
                for (n=0;n<3;n++) u[n] = dimg[n]/rimg;
                for (n=0;n<3;n++) {
                    holder = -((-2.0*chargeprod*alpha*exp(-alpha*alpha*rsq))/(sqrtPI*rimg) - (chargeprod*erfc(alpha*rimg)/rsq))*u[n];
                    af[n] += holder;
                    atomicAdd(&(atom_list[j].f[n]), -holder);                
                }
            }
            // k-space forces
			if (anchoratom.molid < atom_list[j].molid) { // no cutoff for this.
            for (int n=0;n<3;n++) { //x,y,z
              // loop k vectors
				// EWALD k-vectors     
					int l[3],p,q; double k[3] = {0,0,0};
					for (l[0] = 0; l[0] <= kmax; l[0]++) {
						for (l[1] = (!l[0] ? 0 : -kmax); l[1] <= kmax; l[1]++) {
							for (l[2] = ((!l[0] && !l[1]) ? 1 : -kmax); l[2] <= kmax; l[2]++) {
								// skip if norm is out of sphere
								if (l[0]*l[0] + l[1]*l[1] + l[2]*l[2] > kmax*kmax) continue;
								
								/* get reciprocal lattice vectors */				                
								for (p=0; p<3; p++) {
								    for (q=0, k[p] = 0; q < 3; q++) {
								        k[p] += 2.0*M_PI*reciprocal_basis[3*q+p] * l[q];
								    }
								}
								k_sq = k[0]*k[0] + k[1]*k[1] + k[2]*k[2];
                    holder = chargeprod*invV*fourPI*k[n]*
                        exp(-k_sq/(4*alpha*alpha))*
                        sin(k[0]*dimg[0]+
                            k[1]*dimg[1]+
                            k[2]*dimg[2])/k_sq;
                    af[n] += holder;
                    atomicAdd(&(atom_list[j].f[n]), -holder);
								
								
							} // end for l[2], n
						} // end for l[1], m
					} // end for l[0], l
	            } // end 3D
            } // end k-space if not duplicate

            } // end pair loop j for anchoratom

            // finally add ES contribution to anchor-atom
            for (n=0;n<3;n++) atomicAdd(&(atom_list[i].f[n]), af[n]);
        } // end ES component

        //if (i==0) printf("COUNT: %i\n",count);
    } // end if i<n (all threads)
}


__global__
void calculateForceNopbcKernel(cuda_atom * atom_list, int N, int pformD) {
    // define thread id
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // only run for real atoms (no ghost threads)
    if(i<N){   
        const register cuda_atom anchoratom = atom_list[i];
        //printf("I AM THREAD %i\n", i);
        //atom_list[i].pos[0] += cutoff;
       const int pform = pformD;
        const double cutoff=10.; // default 10 A for no-pbc cutoff.
        double d[3], r,r2;
        int j,n;
        double sig,eps,r6,s6,u[3]={0,0,0};
        //int count=0;
        register double af[3] = {0,0,0}; // accumulated forces for anchoratom
        double holder,chargeprod; // for ES force    
        //printf("basis[3] = %f\n", basis[3]);
        __syncthreads();
        // order N instead of N^2 bc this runs on all GPU cores at once (basically)

        // if LJ 
        if (pform == 0 || pform == 1) {
        for (j=i+1;j<N;j++) {

           if (anchoratom.molid == atom_list[j].molid) continue; // skip same molecule 
            if (anchoratom.frozen && atom_list[j].frozen) continue; // skip frozens            

           
            // get R (nearest image)
            for (n=0;n<3;n++) d[n] = anchoratom.pos[n] - atom_list[j].pos[n];
            r2=0;
            for (n=0;n<3;n++) {
                r2 += d[n]*d[n];
            }
            r = sqrt(r2);
               
                if (r <= cutoff) {
           
                 sig = anchoratom.sig;
                if (sig != atom_list[j].sig) sig = 0.5*(sig+atom_list[j].sig);
                eps = anchoratom.eps;
                if (eps != atom_list[j].eps) eps = sqrt(eps * atom_list[j].eps);

                if (sig == 0 || eps == 0) continue;
                
                r6 = r2*r2*r2;
                s6 = sig*sig;
                s6 *= s6 * s6;
        
                    for (n=0;n<3;n++) {
                        holder = 24.0*d[n]*eps*(2*(s6*s6)/(r6*r6*r2) - s6/(r6*r2));
                        atomicAdd(&(atom_list[j].f[n]), -holder); 
                        af[n] += holder;      
                    }
                }

        } // end pair j
        
        // finally add the accumulated forces (stored on register) to the anchor atom
        for (n=0;n<3;n++)
            atomicAdd(&(atom_list[i].f[n]), af[n]);
        
        } // end if LJ
        // ==============================================================================
        // Now handle electrostatics
        if (pform == 1) {
            for (n=0;n<3;n++) af[n]=0; // reset register-stored force for anchoratom.
           for (j=i+1;j<N;j++) {
                if (anchoratom.frozen && atom_list[j].frozen) continue; // don't do frozen pairs
                if (anchoratom.charge == 0 || atom_list[j].charge == 0) continue; // skip 0-force
                if (anchoratom.molid == atom_list[j].molid) continue; // don't do molecule with itself

               // get R (nearest image)
            for (n=0;n<3;n++) d[n] = anchoratom.pos[n] - atom_list[j].pos[n];
            r2=0;
            for (n=0;n<3;n++) {
                r2 += d[n]*d[n];
            }
            r = sqrt(r2);

            if (r <= cutoff)  { //&& (anchoratom.molid < atom_list[j].molid)) { // non-duplicated pairs, not intramolecular, not beyond cutoff
                chargeprod = anchoratom.charge * atom_list[j].charge;
                for (n=0;n<3;n++) u[n] = d[n]/r;
                for (n=0;n<3;n++) {
                    holder = chargeprod/r2 * u[n];
                    af[n] += holder;
                    atomicAdd(&(atom_list[j].f[n]), -holder);                
                }
            }

            } // end pair loop j 

            // finally add ES contribution to anchor-atom
            for (n=0;n<3;n++) atomicAdd(&(atom_list[i].f[n]), af[n]);
        } // end ES component

    } // end if i<n (all threads)
} // end no-pbc force


/*
__global__
void velocityVerletKernel(cuda_molecule * molecule_list, int N, int md_mode) {
    // define thread id
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    register cuda_molecule anchormolecule = molecule_list[i];

    // I ain't afraid o' no ghost 
    if(i<N){
            



    }
}



void CUDA_verlet(System &system) {
    const int N = (int)system.stats.count_movables;
    const int block_size = system.constants.cuda_block_size;
    const int molecules_array_size=sizeof(cuda_molecule)*N;
    cuda_atom H[N]; // host atoms
    cuda_atom *D; // device atoms (gpu)
    for (int i=0;i<system.molecules.size();i++) {
        for (int n=0;n<3;n++) {
            H[i].old_acc[n] = system.molecules[i].old_acc[n];
            H[i].acc[n] = system.molecules[i].acc[n];
            H[i].old_ang_acc[n] = system.molecules[i].old_ang_acc[n];
            H[i].ang_acc[n] = system.molecules[i].ang_acc[n];
            H[i].vel[n] = system.molecules[i].vel[n];
            H[i].ang_vel[n] = system.molecules[i].ang_vel[n];
            
        }
    }

    // allocate memory on GPU
    hipMalloc((void**) &D, molecules_array_size);
    hipMemcpy(D, H, molecules_array_size, hipMemcpyHostToDevice);

    // grid elements
    int dimGrid = ceil((double)N/block_size);
    int dimBlock = block_size;

    // determine molecular or atomic motion
    int md_mode = system.constants.md_mode;
    if (md_mode == MD_ATOMIC) md_mode = 0;
    else if (md_mode == MD_MOLECULAR) md_mode = 1;

    velocityVerletKernel<<< dimGrid, dimBlock >>>(D,N,md_mode);
    // make sure the threads are synced so we don't overflow
    hipDeviceSynchronize();
    // copy device data back to host
    hipMemcpy(H, D, molecules_array_size, hipMemcpyDeviceToHost);

    index=0;
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            for (int n=0;n<3;n++) {
                system.molecules[i].atoms[j].force[n] = H[index].f[n];
            }
            index++;
        }
    }

    //printf("H[0] force = %f %f %f\n",system.molecules[0].atoms[0].force[0], system.molecules[0].atoms[0].force[1], system.molecules[0].atoms[0].force[2]);


     hipFree(D);



}
*/

void CUDA_force(System &system) {

    const int N = (int)system.constants.total_atoms;
    const int block_size = system.constants.cuda_block_size; 
    const int atoms_array_size=sizeof(cuda_atom)*N;
    int index=0;

    cuda_atom H[N]; // host atoms
    cuda_atom *D; // device atoms (gpu)
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            H[index].molid = i;
            H[index].sig = system.molecules[i].atoms[j].sig;
            H[index].eps = system.molecules[i].atoms[j].eps;
            H[index].charge = system.molecules[i].atoms[j].C;
            for (int n=0;n<3;n++) {
                H[index].pos[n] = system.molecules[i].atoms[j].pos[n];       
                H[index].f[n] = 0; // initialize to zero
            }
            H[index].frozen = system.molecules[i].atoms[j].frozen;     
            index++;       
        }
    }

    int bs = sizeof(double)*9;
    double *basis;
    double *reciprocal_basis;
    basis = (double*)malloc(bs);
    reciprocal_basis = (double*)malloc(bs);
    double *dbasis;
    double *dreciprocal_basis;

    for (int p=0;p<3;p++) {
        for (int q=0;q<3;q++) {
            basis[3*q+p] = system.pbc.basis[p][q]; // quite sure correct.
            reciprocal_basis[3*q+p] = system.pbc.reciprocal_basis[p][q]; // quite sure correct.
        }
    }
    //system.pbc.printBasis();

    

    //for (int l=0;l<9;l++) printf("basis[%i] = %f\n", l,basis[l]);

    // allocate memory on GPU
    hipMalloc((void**) &dbasis, bs);
    hipMemcpy(dbasis, basis, bs, hipMemcpyHostToDevice);
    hipMalloc((void**) &dreciprocal_basis, bs);
    hipMemcpy(dreciprocal_basis, reciprocal_basis, bs, hipMemcpyHostToDevice); 
    hipMalloc((void**) &D, atoms_array_size);
    hipMemcpy(D, H, atoms_array_size, hipMemcpyHostToDevice);
	//hipMalloc((void**) &Dkvecs, kvecs_size);
	//hipMemcpy(Dkvecs, Hkvecs, kvecs_size, hipMemcpyHostToDevice);

    // grid elements
    int dimGrid = ceil((double)N/block_size);
    int dimBlock = block_size;   

    // assign potential form for force calculator
    int pform,theval=system.constants.potential_form;
    if (theval == POTENTIAL_LJ || theval == POTENTIAL_LJES || theval == POTENTIAL_LJESPOLAR)
        pform=0;
    if (theval == POTENTIAL_LJES || theval == POTENTIAL_LJESPOLAR)
        pform=1;

    calculateForceKernel<<< dimGrid, dimBlock >>>(D,N,system.pbc.cutoff, dbasis, dreciprocal_basis, pform, system.constants.ewald_alpha, system.constants.ewald_num_k, system.constants.ewald_kmax);
    // make sure the threads are synced so we don't overflow
    hipDeviceSynchronize();
    // copy device data back to host
    hipMemcpy(H, D, atoms_array_size, hipMemcpyDeviceToHost);

    //for (int i=0;i<N;i++) printf("H[%i] force0 = %f\n", i, H[i].f[0]);
    index=0;
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            for (int n=0;n<3;n++) {
                system.molecules[i].atoms[j].force[n] = H[index].f[n];
            }     
            index++;       
        }
    }

    //printf("H[0] force = %f %f %f\n",system.molecules[0].atoms[0].force[0], system.molecules[0].atoms[0].force[1], system.molecules[0].atoms[0].force[2]);
 

    // clean up -- so we don't have a memory leak
     hipFree(D);
     hipFree(dbasis);
     hipFree(dreciprocal_basis);
     free(basis);
     free(reciprocal_basis);


    // we're done. forces have been calc'd on GPU and written to local mem.
}

void CUDA_force_nopbc(System &system) {

    const int N = (int)system.constants.total_atoms;
    const int block_size = system.constants.cuda_block_size; 
    const int atoms_array_size=sizeof(cuda_atom)*N;
    int index=0;

    cuda_atom H[N]; // host atoms
    cuda_atom *D; // device atoms (gpu)
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            H[index].molid = i;
            H[index].sig = system.molecules[i].atoms[j].sig;
            H[index].eps = system.molecules[i].atoms[j].eps;
            H[index].charge = system.molecules[i].atoms[j].C;
            for (int n=0;n<3;n++) {
                H[index].pos[n] = system.molecules[i].atoms[j].pos[n];       
                H[index].f[n] = 0; // initialize to zero
            }
            H[index].frozen = system.molecules[i].atoms[j].frozen;     
            index++;       
        }
    }

    // allocate memory on GPU
    hipMalloc((void**) &D, atoms_array_size);
    hipMemcpy(D, H, atoms_array_size, hipMemcpyHostToDevice);

    // grid elements
    int dimGrid = ceil((double)N/block_size);
    int dimBlock = block_size;   

    // assign potential form for force calculator
    int pform,theval=system.constants.potential_form;
    if (theval == POTENTIAL_LJ || theval == POTENTIAL_LJES || theval == POTENTIAL_LJESPOLAR)
        pform=0;
    if (theval == POTENTIAL_LJES || theval == POTENTIAL_LJESPOLAR)
        pform=1;

    calculateForceNopbcKernel<<< dimGrid, dimBlock >>>(D,N, pform);
    // make sure the threads are synced so we don't overflow
    hipDeviceSynchronize();
    // copy device data back to host
    hipMemcpy(H, D, atoms_array_size, hipMemcpyDeviceToHost);

    //for (int i=0;i<N;i++) printf("H[%i] force0 = %f\n", i, H[i].f[0]);
    index=0;
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            for (int n=0;n<3;n++) {
                system.molecules[i].atoms[j].force[n] = H[index].f[n];
            }     
            index++;       
        }
    }

     hipFree(D);

    // we're done. forces have been calc'd on GPU and written to local mem.
}
