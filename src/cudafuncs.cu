#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

/* this is an explicit definition for atomicAdd, to be safe */
__device__ double atomicAdd(double* address, double val)
{
 unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do { assumed = old;
  old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed))); // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) 
  }
  while (assumed != old);
  return __longlong_as_double(old);
}

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {

        y[i] = a*x[i] + y[i];
        printf("y[%i] = %f\n", i, y[i]);
    }
}

void CUDA_force(System &system) {

    // first send information to the device
    int system_size = sizeof(system);
    printf("SYSTEM SIZE: %i\n", system_size); 

    int atoms_size = system.constants.total_atoms * sizeof(system.molecules[0].atoms[0]);
    printf("molecules vector size: %i\n", atoms_size);

    printf("sizeof(Atom): %i\n", (int)sizeof(Atom));
}
