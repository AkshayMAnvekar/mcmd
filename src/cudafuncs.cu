#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


/* this is an explicit definition for atomicAdd, to be safe */
__device__ double atomicAdd(double* address, double val)
{
 unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do { assumed = old;
  old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed))); // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) 
  }
  while (assumed != old);
  return __longlong_as_double(old);
}


// minimal data to send to GPU. this is all that's needed to calc forces.
typedef struct atom_t {
    double pos[3]={0,0,0};
    double eps=0; // lj
    double sig=0; // lj
    double charge=0;
    double f[3]={0,0,0}; // force
    int molid=0;
    int frozen=0;
} cuda_atom;

/*
// same but for molecule
typedef struct molecule_t {
    double old_ang_acc[3]={0,0,0};
    double ang_acc[3]={0,0,0};
    double ang_vel[3]={0,0,0};
    double ang_pos[3]={0,0,0};
    double torque[3]={0,0,0};
    double inertia=0;
    double mass=0;
    double old_acc[3]={0,0,0};
    double acc[3]={0,0,0};
    double vel[3]={0,0,0};
    double com[3]={0,0,0};
    double force[3]={0,0,0};
} cuda_molecule;
*/

__global__
void calculateForceKernel(cuda_atom * atom_list, int N, double cutoff, double * basis, double * reciprocal_basis, int pform, double ewald_alpha) {
    // define thread id
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    register cuda_atom anchoratom = atom_list[i];

    // only run for real atoms (no ghost threads)
    if(i<N){   
        //printf("I AM THREAD %i\n", i);
        //atom_list[i].pos[0] += cutoff;
        const double alpha = ewald_alpha;
        register double rimg, rsq;
        const double sqrtPI=sqrt(M_PI);
        double d[3], di[3], img[3], dimg[3],r,r2,ri,ri2;
        int p,q,j,n;
        double sig,eps,r6,s6,f[3]={0,0,0},u[3]={0,0,0};
        //int count=0;
        register double af[3] = {0,0,0}; // accumulated forces for anchoratom
        double holder,erfc_term,chargeprod; // for ES force    
        //printf("basis[3] = %f\n", basis[3]);
        __syncthreads();
        // order N instead of N^2 bc this runs on all GPU cores at once (basically)

        // if LJ 
        if (pform == 0 || pform == 1) {
        for (j=i+1;j<N;j++) {

           if (anchoratom.molid == atom_list[j].molid) continue; // skip same molecule 
            // get R (nearest image)
            for (n=0;n<3;n++) d[n] = anchoratom.pos[n] - atom_list[j].pos[n];
            for (p=0;p<3;p++) {
                img[p]=0;
                for (q=0;q<3;q++) {
                    img[p] += reciprocal_basis[p*3+q]*d[q];
                    //if (i==0 && j==1188) printf("img[%i] = reciprocal_basis[%i]*d[%i] = %f\n",p,p*3+q,q,reciprocal_basis[p*3+q]*d[q]);
                }
                img[p] = rint(img[p]);
            }
            for (p=0;p<3;p++) {
                di[p] = 0;
                for (q=0;q<3;q++) {
                    di[p] += basis[p*3+q]*img[q];
                }
            }
            for (p=0;p<3;p++) di[p] = d[p] - di[p];
            r2=0;ri2=0;
            for (p=0;p<3;p++) {
                r2 += d[p]*d[p];
                ri2 += di[p]*di[p];
            }
            r = sqrt(r2);
            ri = sqrt(ri2);
            if (ri != ri) {
                rimg=r;
                for (p=0;p<3;p++) dimg[p] = d[p];
            } else {
                rimg=ri;
                for (p=0;p<3;p++) dimg[p] = di[p];
            }
            // distance is now rimg
               
            //if (i==0) {
              //  printf("r[%i].%i = %f\n", i,j,rimg);
                //printf("CUTOFF: %f\n", cutoff);
                //for (int h=0;h<9;h++) {
                  //  printf("basis[%i] = %f\n", h, basis[h]);
                //}
            //}
            
            rsq=rimg*rimg;

                //if (i==0) printf("hi\n");

                sig = anchoratom.sig;
                if (sig != atom_list[j].sig) sig = 0.5*(sig+atom_list[j].sig);
                eps = anchoratom.eps;
                if (eps != atom_list[j].eps) eps = sqrt(anchoratom.eps * atom_list[j].eps);

                if (sig == 0 || eps == 0) continue;

                r6 = rsq*rsq*rsq;
                s6 = sig*sig;
                s6 *= s6 * s6;

                if (rimg <= cutoff) {
                    for (n=0;n<3;n++) {
                        f[n] = 24.0*dimg[n]*eps*(2*(s6*s6)/(r6*r6*rsq) - s6/(r6*rsq));
                        atomicAdd(&(atom_list[j].f[n]), -f[n]); 
                        af[n] += f[n];      
                        
                        //af[n] += f[n];
                    }
                    //if (i==0) count++;
                }

        } // end pair j
        
        // finally add the accumulated forces (stored on register) to the anchor atom
        for (n=0;n<3;n++)
            atomicAdd(&(atom_list[i].f[n]), af[n]);
        
        } // end if LJ
        // ==============================================================================
        // Now handle electrostatics
        if (pform == 1) {
            for (n=0;n<3;n++) af[n]=0; // reset register-stored force for anchoratom.
           for (j=0;j<N;j++) {
                if (i==j) continue; // don't do atom with itself
                if (anchoratom.frozen && atom_list[j].frozen) continue; // don't do frozen pairs
                if (anchoratom.charge == 0 && atom_list[j].charge == 0) continue; // skip 0-force

                chargeprod = anchoratom.charge * atom_list[j].charge;
            

               // get R (nearest image)
            for (n=0;n<3;n++) d[n] = anchoratom.pos[n] - atom_list[j].pos[n];
            for (p=0;p<3;p++) {
                img[p]=0;
                for (q=0;q<3;q++) {
                    img[p] += reciprocal_basis[p*3+q]*d[q];
                }
                img[p] = rint(img[p]);
            }
            for (p=0;p<3;p++) {
                di[p] = 0;
                for (q=0;q<3;q++) {
                    di[p] += basis[p*3+q]*img[q];
                }
            }
            for (p=0;p<3;p++) di[p] = d[p] - di[p];
            r2=0;ri2=0;
            for (p=0;p<3;p++) {
                r2 += d[p]*d[p];
                ri2 += di[p]*di[p];
            }
            r = sqrt(r2);
            ri = sqrt(ri2);
            if (ri != ri) {
                rimg=r;
                for (p=0;p<3;p++) dimg[p] = d[p];
            } else {
                rimg=ri;
                for (p=0;p<3;p++) dimg[p] = di[p];
            }

            rsq=rimg*rimg;
            for (n=0;n<3;n++) u[n] = dimg[n]/rimg;

            if (r <= cutoff && (anchoratom.molid < atom_list[j].molid)) { // non-duplicated pairs, not intramolecular, not beyond cutoff
                erfc_term = erfc(alpha*r);
                for (n=0;n<3;n++) {
                    holder = -((-2.0*chargeprod*alpha*exp(-alpha*alpha*r*r))/(sqrtPI*r) - (chargeprod*erfc_term/rsq))*u[n];
                    af[n] += holder;
                    atomicAdd(&(atom_list[j].f[n]), -holder);                
                }
            } else if (anchoratom.molid == atom_list[j].molid && i != j) { // intramolecular interaction
                for (n=0;n<3;n++) {
                    holder = -((chargeprod*erf(alpha*r))/rsq - (2*chargeprod*alpha*exp(-alpha*alpha*r*r)/(sqrtPI*r)))*u[n];
                    af[n] += holder;
                    atomicAdd(&(atom_list[j].f[n]), -holder);
                }
            }

            } // end pair loop j 

            // finally add ES contribution to anchor-atom
            for (n=0;n<3;n++) atomicAdd(&(atom_list[i].f[n]), af[n]);
        } // end ES component

        //if (i==0) printf("COUNT: %i\n",count);
    } // end if i<n (all threads)
}


/*
__global__
void velocityVerletKernel(cuda_molecule * molecule_list, int N, int md_mode) {
    // define thread id
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    register cuda_molecule anchormolecule = molecule_list[i];

    // I ain't afraid o' no ghost 
    if(i<N){
            



    }
}



void CUDA_verlet(System &system) {
    const int N = (int)system.stats.count_movables;
    const int block_size = system.constants.cuda_block_size;
    const int molecules_array_size=sizeof(cuda_molecule)*N;
    cuda_atom H[N]; // host atoms
    cuda_atom *D; // device atoms (gpu)
    for (int i=0;i<system.molecules.size();i++) {
        for (int n=0;n<3;n++) {
            H[i].old_acc[n] = system.molecules[i].old_acc[n];
            H[i].acc[n] = system.molecules[i].acc[n];
            H[i].old_ang_acc[n] = system.molecules[i].old_ang_acc[n];
            H[i].ang_acc[n] = system.molecules[i].ang_acc[n];
            H[i].vel[n] = system.molecules[i].vel[n];
            H[i].ang_vel[n] = system.molecules[i].ang_vel[n];
            
        }
    }

    // allocate memory on GPU
    hipMalloc((void**) &D, molecules_array_size);
    hipMemcpy(D, H, molecules_array_size, hipMemcpyHostToDevice);

    // grid elements
    int dimGrid = ceil((double)N/block_size);
    int dimBlock = block_size;

    // determine molecular or atomic motion
    int md_mode = system.constants.md_mode;
    if (md_mode == MD_ATOMIC) md_mode = 0;
    else if (md_mode == MD_MOLECULAR) md_mode = 1;

    velocityVerletKernel<<< dimGrid, dimBlock >>>(D,N,md_mode);
    // make sure the threads are synced so we don't overflow
    hipDeviceSynchronize();
    // copy device data back to host
    hipMemcpy(H, D, molecules_array_size, hipMemcpyDeviceToHost);

    index=0;
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            for (int n=0;n<3;n++) {
                system.molecules[i].atoms[j].force[n] = H[index].f[n];
            }
            index++;
        }
    }

    //printf("H[0] force = %f %f %f\n",system.molecules[0].atoms[0].force[0], system.molecules[0].atoms[0].force[1], system.molecules[0].atoms[0].force[2]);


     hipFree(D);



}
*/

void CUDA_force(System &system) {

    const int N = (int)system.constants.total_atoms;
    const int block_size = system.constants.cuda_block_size; 
    const int atoms_array_size=sizeof(cuda_atom)*N;
    int index=0;

    cuda_atom H[N]; // host atoms
    cuda_atom *D; // device atoms (gpu)
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            H[index].molid = i;
            H[index].sig = system.molecules[i].atoms[j].sig;
            H[index].eps = system.molecules[i].atoms[j].eps;
            H[index].charge = system.molecules[i].atoms[j].C;
            for (int n=0;n<3;n++) {
                H[index].pos[n] = system.molecules[i].atoms[j].pos[n];       
                H[index].f[n] = 0; // initialize to zero
            }
            H[index].frozen = system.molecules[i].atoms[j].frozen;     
            index++;       
        }
    }

    int bs = sizeof(double)*9;
    double *basis;
    double *reciprocal_basis;
    basis = (double*)malloc(bs);
    reciprocal_basis = (double*)malloc(bs);
    double *dbasis;
    double *dreciprocal_basis;

    for (int p=0;p<3;p++) {
        for (int q=0;q<3;q++) {
            basis[3*q+p] = system.pbc.basis[p][q]; // quite sure correct.
            reciprocal_basis[3*q+p] = system.pbc.reciprocal_basis[p][q]; // quite sure correct.
        }
    }
    //system.pbc.printBasis();

    //for (int l=0;l<9;l++) printf("basis[%i] = %f\n", l,basis[l]);

    // allocate memory on GPU
    hipMalloc((void**) &dbasis, bs);
    hipMemcpy(dbasis, basis, bs, hipMemcpyHostToDevice);
    hipMalloc((void**) &dreciprocal_basis, bs);
    hipMemcpy(dreciprocal_basis, reciprocal_basis, bs, hipMemcpyHostToDevice); 
    hipMalloc((void**) &D, atoms_array_size);
    hipMemcpy(D, H, atoms_array_size, hipMemcpyHostToDevice);

    // grid elements
    int dimGrid = ceil((double)N/block_size);
    int dimBlock = block_size;   

    // assign potential form for force calculator
    int pform,theval=system.constants.potential_form;
    if (theval == POTENTIAL_LJ || theval == POTENTIAL_LJES || theval == POTENTIAL_LJESPOLAR)
        pform=0;
    if (theval == POTENTIAL_LJES || theval == POTENTIAL_LJESPOLAR)
        pform=1;

    calculateForceKernel<<< dimGrid, dimBlock >>>(D,N,system.pbc.cutoff, dbasis, dreciprocal_basis, pform, system.constants.ewald_alpha);
    // make sure the threads are synced so we don't overflow
    hipDeviceSynchronize();
    // copy device data back to host
    hipMemcpy(H, D, atoms_array_size, hipMemcpyDeviceToHost);

    //for (int i=0;i<N;i++) printf("H[%i] force0 = %f\n", i, H[i].f[0]);
    index=0;
    for (int i=0;i<system.molecules.size();i++) {
        for (int j=0;j<system.molecules[i].atoms.size();j++) {
            for (int n=0;n<3;n++) {
                system.molecules[i].atoms[j].force[n] = H[index].f[n];
            }     
            index++;       
        }
    }

    //printf("H[0] force = %f %f %f\n",system.molecules[0].atoms[0].force[0], system.molecules[0].atoms[0].force[1], system.molecules[0].atoms[0].force[2]);
 

     hipFree(D);

    // we're done. forces have been calc'd on GPU and written to local mem.
}
